#include "hip/hip_runtime.h"
#include "project.h"

char *programName;

int main(int argc, char *argv[])
{
  programName = argv[0];
  if (argc < 4)
  {
    printf("Usage:\n");
    printf("    %s tiffDirectory threshold outputFile\n", programName);
    exit(0);
  }
  char *dirName = argv[1];
  unsigned long threshold = atol(argv[2]);
  if (threshold == 0)
  {
    fprintf(stderr, "%s: invalid threshold %s\n", programName, argv[2]);
    exit(0);
  }
  struct dirent **fileList;
  int numFiles = scandir(dirName, &fileList, tiffSelector, alphasort);
  if (numFiles == -1)
  {
    fprintf(stderr, "%s: unable to open directory %s\n", programName, dirName);
    exit(1);
  } else if (numFiles == 0)
  {
    fprintf(stderr, "%s: no TIFF files found in %s\n", programName, dirName);
    exit(2);
  }
  printf("found %d files\n", numFiles);

  // change working directory to directory of TIFF files
  char oldWorkingDirectory[1024];
  getcwd(oldWorkingDirectory, sizeof(oldWorkingDirectory));
  chdir(dirName);

  Volume volume;
  TIFF *slice = TIFFOpen(fileList[0]->d_name, "r");
  if (slice == NULL)
  {
    fprintf(stderr, "%s: unable to open TIFF file %s\n", programName, fileList[0]->d_name);
    exit(3);
  }
  TIFFGetField(slice, TIFFTAG_IMAGEWIDTH, &(volume.width));
  TIFFGetField(slice, TIFFTAG_IMAGELENGTH, &(volume.height));
  volume.depth = numFiles;
  if (!TIFFGetField(slice, TIFFTAG_SAMPLEFORMAT, &(volume.pixelFormat)))
  {
    volume.pixelFormat = SAMPLEFORMAT_UINT;
  }
  int bitsPerSample = 0;
  TIFFGetField(slice, TIFFTAG_BITSPERSAMPLE, &bitsPerSample);
  volume.bytesPerPixel = bitsPerSample / 8;
  mallocVolume(&volume);
  TIFFClose(slice);
  printf("we are dealing with a %dx%dx%d volume with %d bytes per pixel\n", volume.width, volume.height, volume.depth, volume.bytesPerPixel);

  // now read each slice
  int z;
  char *buf = volume.data;
  for (z = 0; z < numFiles; z++)
  {
    slice = TIFFOpen(fileList[z]->d_name, "r");
    if (slice == NULL)
    {
      fprintf(stderr, "%s: unable to open TIFF file %s\n", programName, fileList[z]->d_name);
      exit(3);
    }
    int numStrips = TIFFNumberOfStrips(slice);
    int strip;
    int bytesRead;
    for (strip = 0; strip < numStrips; strip++)
    {
      bytesRead = TIFFReadEncodedStrip(slice, strip, buf, (tsize_t)-1);
      if (bytesRead == -1)
      {
        fprintf(stderr, "%s: error reading tiff file %s\n", programName, fileList[z]->d_name);
        exit(3);
      }
      buf += bytesRead;
    }
    TIFFClose(slice);
  }
  chdir(oldWorkingDirectory);
  printf("loaded images successfully\n");
  // compute some statistics about the volume
  printf("min: %lu; max: %lu\n", minIntensity(&volume), maxIntensity(&volume));

  // compute the thresholded volume
  Volume tVol;
  tVol.width = volume.width;
  tVol.height = volume.height;
  tVol.depth = volume.depth;
  tVol.bytesPerPixel = 1;
  mallocVolume(&tVol);
  int x, y;
  for (z = 0; z < volume.depth; z++)
  {
    for (y = 0; y < volume.height; y++)
    {
      for (x = 0; x < volume.width; x++)
      {
        setIntensity(&tVol, x, y, z, getIntensity(&volume, x, y, z) >= threshold ? 255 : 0);
      }
    }
  }
  printf("min: %lu; max: %lu\n", minIntensity(&tVol), maxIntensity(&tVol));
  writeRaw(&tVol, argv[3]);

  free(fileList);
  _TIFFfree(volume.data);
}
