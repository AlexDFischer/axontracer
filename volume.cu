#include "hip/hip_runtime.h"
#include "project.h"

unsigned long getIntensity(Volume *volume, int x, int y, int z)
{
  size_t index = z * volume->width * volume->height + y * volume->width + x;
  unsigned long result = 0;
  switch (volume->bytesPerPixel)
  {
    case 1:
      return ((uint8_t *) (volume->data))[index];
    case 2:
      return ((uint16_t *) (volume->data))[index];
    case 4:
      return ((uint32_t *) (volume->data))[index];
    case 8:
      return ((uint64_t *) (volume->data))[index];
  }
  return result;
}

void setIntensity(Volume *volume, int x, int y, int z, unsigned long intensity)
{
  size_t index = z * volume->width * volume->height + y * volume->width + x;
  switch (volume->bytesPerPixel)
  {
    case 1:
      ((uint8_t *) (volume->data))[index] = intensity;
      break;
    case 2:
      ((uint16_t *) (volume->data))[index] = intensity;
      break;
    case 4:
      ((uint32_t *) (volume->data))[index] = intensity;
      break;
    case 8:
      ((uint64_t *) (volume->data))[index] = intensity;
      break;
  }
}

void cudaMallocManagedVolume(Volume *volume)
{
  hipMallocManaged(&(volume->data), volume->width * volume->height * volume->depth * volume->bytesPerPixel);
}

void _TIFFMallocVolume(Volume *volume)
{
  volume->data = (char *) _TIFFmalloc(volume->width * volume->height * volume->depth * volume->bytesPerPixel);
}

void mallocVolume(Volume *volume)
{
  volume->data = (char *) malloc(volume->width * volume->height * volume->depth * volume->bytesPerPixel);
}

unsigned long maxIntensity(Volume *volume)
{
  int x, y, z;
  unsigned long max = getIntensity(volume, 0, 0, 0);
  for (z = 0; z < volume->depth; z++)
  {
    for (y = 0; y < volume->height; y++)
    {
      for (x = 0; x < volume->width; x++)
      {
        unsigned long val = getIntensity(volume, x, y, z);
        if (val > max)
        {
          max = val;
        }
      }
    }
  }
  return max;
}

unsigned long minIntensity(Volume *volume)
{
  int x, y, z;
  unsigned long min = getIntensity(volume, 0, 0, 0);
  for (z = 0; z < volume->depth; z++)
  {
    for (y = 0; y < volume->height; y++)
    {
      for (x = 0; x < volume->width; x++)
      {
        unsigned long val = getIntensity(volume, x, y, z);
        if (val < min)
        {
          min = val;
        }
      }
    }
  }
  return min;
}

void printVolume(Volume *volume)
{
  int x, y, z;
  for (z = 0; z < volume->depth; z++)
  {
    printf("SLICE %d\n", z);
    for (y = 0; y < volume->height; y++)
    {
      for (x = 0; x < volume->width; x++)
      {
        printf("%02lx ", getIntensity(volume, x, y, z));
      }
      printf("\n");
    }
  }
}

/**
 * Reads the given RAW file into the given volume. volume->data is
 * hipMallocManaged and should be cudaFreed when done. fileName should not have
 * an extension, as the RAW and TXT extensions will be added onto it.
 */
int readRaw(Volume *volume, char *fileName)
{
  int len = strlen(fileName);
  char *fileNameExt = (char *) malloc(len + 5);
  strcpy(fileNameExt, fileName);
  strcpy(fileNameExt + len, ".txt");
  FILE *f = fopen(fileNameExt, "r");
  if (f == NULL)
  {
    fprintf(stderr, "%s: unable to open file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  if (fscanf(f, "%dx%dx%d\n", &(volume->width), &(volume->height), &(volume->depth)) != 3)
  {
    fprintf(stderr, "%s: invalid first line of %s\n", programName, fileNameExt);
    return -1;
  }
  if (fscanf(f, "%d\n", &(volume->bytesPerPixel)) != 1)
  {
    fprintf(stderr, "%s: invalid second line of %s\n", programName, fileNameExt);
    return -1;
  }
  int scaleX, scaleY, scaleZ;
  if (fscanf(f, "scale: %d:%d:%d", &scaleX, &scaleY, &scaleZ) != 3)
  {
    fprintf(stderr, "%s: invalid third line of %s\n", programName, fileNameExt);
    return -1;
  }
  if (fclose(f))
  {
    fprintf(stderr, "%s: unable to close file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  cudaMallocManagedVolume(volume);
  strcpy(fileNameExt + len, ".raw");
  f = fopen(fileNameExt, "r");
  fread(volume->data, volume->bytesPerPixel, volume->width * volume->height * volume->depth, f);
  if (ferror(f))
  {
    fprintf(stderr, "%s: error reading from file %s: %s\n", programName, fileNameExt, strerror(ferror(f)));
    return -1;
  }
  free(fileNameExt);
  return 0;
}

/**
 * Writes the given volume to the given RAW file. fileName should not have an
 * extension, as the RAW and TXT extensions will be added onto it.
 */
int writeRaw(Volume *volume, char *fileName)
{
  int len = strlen(fileName);
  char *fileNameExt = (char *) malloc(len + 5);
  strcpy(fileNameExt, fileName);
  strcpy(fileNameExt + len, ".txt");
  FILE *f = fopen(fileNameExt, "w");
  if (f == NULL)
  {
    fprintf(stderr, "%s: unable to open file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  fprintf(f, "%dx%dx%d\n", volume->width, volume->height, volume->depth);
  fprintf(f, "%d\n", volume->bytesPerPixel);
  fprintf(f, "scale: 1:1:1\n");
  if (ferror(f))
  {
    fprintf(stderr, "%s: error writing to file %s: %s\n", programName, fileNameExt, strerror(ferror(f)));
    return -1;
  }
  fclose(f);
  strcpy(fileNameExt + len, ".raw");
  f = fopen(fileNameExt, "w");
  if (f == NULL)
  {
    fprintf(stderr, "%s: unable to open file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  fwrite(volume->data, volume->bytesPerPixel, volume->width * volume->height * volume->depth, f);
  if (ferror(f))
  {
    fprintf(stderr, "%s: error writing to file %s: %s\n", programName, fileNameExt, strerror(ferror(f)));
    return -1;
  }
  fclose(f);
  free(fileNameExt);
  return 0;
}
