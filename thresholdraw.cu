#include "hip/hip_runtime.h"
#include "project.h"

char *programName;

__device__
unsigned long deviceGetIntensity(Volume *volume, int x, int y, int z)
{
  size_t index = z * volume->width * volume->height + y * volume->width + x;
  unsigned long result = 0;
  switch (volume->bytesPerPixel)
  {
    case 1:
      return ((uint8_t *) (volume->data))[index];
    case 2:
      return ((uint16_t *) (volume->data))[index];
    case 4:
      return ((uint32_t *) (volume->data))[index];
    case 8:
      return ((uint64_t *) (volume->data))[index];
  }
  return result;
}

__device__
void deviceSetIntensity(Volume *volume, int x, int y, int z, unsigned long intensity)
{
  size_t index = z * volume->width * volume->height + y * volume->width + x;
  switch (volume->bytesPerPixel)
  {
    case 1:
      ((uint8_t *) (volume->data))[index] = intensity;
      break;
    case 2:
      ((uint16_t *) (volume->data))[index] = intensity;
      break;
    case 4:
      ((uint32_t *) (volume->data))[index] = intensity;
      break;
    case 8:
      ((uint64_t *) (volume->data))[index] = intensity;
      break;
  }
}

__global__
void cudaThreshold(Volume *vol1, Volume *vol2, int n, unsigned long threshold, unsigned long low, unsigned long high)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  for (int i = index; i < n; i += stride)
  /*
  int voxelsPerThread = (n + blockDim.x * gridDim.x - 1) / (blockDim.x * gridDim.x);
  int index = (blockIdx.x * blockDim.x + threadIdx.x) * voxelsPerThread;
  int endIndex = min(index + voxelsPerThread, n);
  for (int i = index; i < endIndex; i++)
  */
  {
    if (deviceGetIntensity(vol1, i, 0, 0) < threshold)
    {
      deviceSetIntensity(vol2, i, 0, 0, low);
    } else
    {
      deviceSetIntensity(vol2, i, 0, 0, high);
    }
  }
}

int main(int argc, char *argv[])
{
  programName = argv[0];
  if (argc < 4)
  {
    printf("Usage:\n");
    printf("    %s rawInput rawOutput threshold\n", programName);
    exit(0);
  }
  unsigned long threshold = atol(argv[3]);
  if (threshold == 0)
  {
    fprintf(stderr, "%s: invalid threshold: %s\n", programName, argv[3]);
    exit(0);
  }
  Volume *volume;
  hipMallocManaged(&volume, sizeof(Volume));
  if (readRaw(volume, argv[1]) == -1)
  {
    exit(0);
  }
  Volume *tVol;
  hipMallocManaged(&tVol, sizeof(Volume));
  tVol->width = volume->width;
  tVol->height = volume->height;
  tVol->depth = volume->depth;
  tVol->bytesPerPixel = 1;
  cudaMallocManagedVolume(tVol);
  struct hipDeviceProp_t prop;
  int device, deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("device count: %d\n", deviceCount);
  for (device = 0; device < deviceCount; device++)
  {
    //hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    printf("DEVICE NO. %d:\n", device);
    printf("    Name: %s, supports compute capability %d.%d.\n", prop.name, prop.major, prop.minor);
    printf("    This device can support %d threads per block, and it has %d multiprocessors.\n", prop.maxThreadsPerBlock, prop.multiProcessorCount);
  }
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  int numVoxels = volume->width * volume->height * volume->depth;
  int threadsPerBlock = prop.maxThreadsPerBlock;
  int numBlocks = (numVoxels + threadsPerBlock - 1) / threadsPerBlock;
  numBlocks = 24; // empirically found to be the best. I wonder why?
  printf("We will be using device number %d. We will be using %d blocks of %d threads each for a total of %d threads to threshold %d voxels\n", device, numBlocks, threadsPerBlock, numBlocks * threadsPerBlock, numVoxels);
  printf("Each thread will threshold at most %d voxels.\n", (numVoxels + numBlocks * threadsPerBlock - 1) / (numBlocks * threadsPerBlock));
  cudaThreshold<<<numBlocks, threadsPerBlock>>>(volume, tVol, numVoxels, threshold, 0, 255);
  hipDeviceSynchronize();
  writeRaw(tVol, argv[2]);
  printf("wrote to raw file %s\n", argv[2]);
  hipFree(volume->data);
  hipFree(tVol->data);
  hipFree(volume);
  hipFree(tVol);
}
